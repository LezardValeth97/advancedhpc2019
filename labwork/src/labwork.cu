#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    // do something here
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices = 0;
    // get all devices
    hipGetDeviceCount(&nDevices);
    printf("Number total of GPU : %d\n\n", nDevices);
    for (int i = 0; i < nDevices; i++){
        // get informations from individual device
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        // something more here
    }

}

void Labwork::labwork3_GPU() {
    // Calculate number of pixels

    // Allocate CUDA memory    

    // Copy CUDA Memory from CPU to GPU

    // Processing

    // Copy CUDA Memory from GPU to CPU

    // Cleaning
}

void Labwork::labwork4_GPU() {
}

void Labwork::labwork5_GPU(bool shared) {
}

void Labwork::labwork6_GPU() {
}

void Labwork::labwork7_GPU() {
}

void Labwork::labwork8_GPU() {
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU(){
}


























