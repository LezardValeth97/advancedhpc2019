#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <time.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            //labwork.labwork5_CPU();
            //labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    #pragma omp parallel for schedule(dynamic)
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices = 0;
    // get all devices
    hipGetDeviceCount(&nDevices);
    printf("Number total of GPU : %d\n\n", nDevices);
    for (int i = 0; i < nDevices; i++){
        // get informations from individual device
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        // something more here
        printf("GPU #%d\n", i);
        printf("GPU name: %s\n", prop.name);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Number of cores: %d\n", getSPcores(prop));
        printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
        printf("Warp Size: %d\n", prop.warpSize);
        printf("Memory Clock Rate: %d\n", prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\nDevices", prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

}

// Write a grayscale kernel
__global__ void grayscale(uchar3 *input, uchar3 *output) {
    // this will execute in a device core
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    output[tid].x = (input[tid].x + input[tid].y +input[tid].z) / 3;
    output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork3_GPU() {
    // Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
 
    // Allocate CUDA memory
    uchar3 *devInput;
    uchar3 *devOutput;
    hipMalloc(&devInput, pixelCount *sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount *sizeof(uchar3));

    // Copy InputImage from CPU (host) to GPU (device)
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);

    // Processing : launch the kernel
    int blockSize = 1024;
    int numBlock = pixelCount / blockSize;  
    grayscale<<<numBlock, blockSize>>>(devInput, devOutput);

    // Copy CUDA Memory from GPU to CPU
    // allocate memory for the output on the host
    outputImage = static_cast<char *>(malloc(pixelCount * sizeof(uchar3)));  
    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);   

    // Cleaning
    hipFree(devInput);
    hipFree(devOutput);
}

// Write a grayscale kernel
__global__ void grayscaleVer2D(uchar3 *input, uchar3 *output, int width, int height) {
    // this will execute in a device core
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;

    int tid = tidx + tidy * width;
    output[tid].x = (input[tid].x + input[tid].y +input[tid].z) / 3;
    output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork4_GPU() {
    // Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
 
    // Allocate CUDA memory
    uchar3 *devInput;
    uchar3 *devOutput;
    hipMalloc(&devInput, pixelCount *sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount *sizeof(uchar3));

    // Copy InputImage from CPU (host) to GPU (device)
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);

    // Processing : launch the kernel
    // int blockSize = 1024;
    // int numBlock = pixelCount / blockSize;  
    // grayscale<<<numBlock, blockSize>>>(devInput, devOutput);
    dim3 blockSize = dim3(32, 32);
    //dim3 gridSize = dim3(8, 8);
    dim3 gridSize = dim3((inputImage->width + blockSize.x -1) / blockSize.x, (inputImage->height + blockSize.y -1) / blockSize.y);
    grayscaleVer2D<<<gridSize, blockSize>>>(devInput, devOutput, inputImage->width, inputImage->height);

    // Copy CUDA Memory from GPU to CPU
    // allocate memory for the output on the host
    outputImage = static_cast<char *>(malloc(pixelCount * sizeof(uchar3)));  
    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);   

    // Cleaning
    hipFree(devInput);
    hipFree(devOutput);
}

void Labwork::labwork5_GPU() {
}

void Labwork::labwork6_GPU() {
}

void Labwork::labwork7_GPU() {
}

void Labwork::labwork8_GPU() {
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU(){
}


























