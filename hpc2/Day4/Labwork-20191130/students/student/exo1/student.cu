#include "hip/hip_runtime.h"
#include "student.h"

namespace {
	__device__
	float3 RGB2HSV( const uchar3 inRGB ) {
		const float R = (float)( inRGB.x ) / 256.f;
		const float G = (float)( inRGB.y ) / 256.f;
		const float B = (float)( inRGB.z ) / 256.f;
	
		const float min		= fminf( R, fminf( G, B ) );
		const float max		= fmaxf( R, fmaxf( G, B ) );
		const float delta	= max - min;
	
		// H
		float H;
		if		( delta < FLT_EPSILON )  
			H = 0.f;
		else if	( max == R )	
			H = 60.f * ( G - B ) / ( delta + FLT_EPSILON )+ 360.f;
		else if ( max == G )	
			H = 60.f * ( B - R ) / ( delta + FLT_EPSILON ) + 120.f;
		else					
			H = 60.f * ( R - G ) / ( delta + FLT_EPSILON ) + 240.f;
		while	( H >= 360.f )	
			H -= 360.f ;
	
		// S
		const float S = max < FLT_EPSILON ? 0.f : 1.f - min / max;
	
		// V
		const float V = max;
	
		return make_float3( H, S, V );
	}
	
	__device__
	uchar3 HSV2RGB( const float H, const float S, const float V ) {
		const float	d	= H / 60.f;
		const int	hi	= (int)d % 6;
		const float f	= d - (float)hi;
	
		const float l   = V * ( 1.f - S );
		const float m	= V * ( 1.f - f * S );
		const float n	= V * ( 1.f - ( 1.f - f ) * S );
	
		float R, G, B;
	
		if		( hi == 0 ) 
			{ R = V; G = n;	B = l; }
		else if ( hi == 1 ) 
			{ R = m; G = V;	B = l; }
		else if ( hi == 2 ) 
			{ R = l; G = V;	B = n; }
		else if ( hi == 3 ) 
			{ R = l; G = m;	B = V; }
		else if ( hi == 4 ) 
			{ R = n; G = l;	B = V; }
		else				
			{ R = V; G = l;	B = m; }
			  
		return make_uchar3( R * 256.f, G * 256.f, B * 256.f );
	}
	
}


// ============================================ Exercise 1
// Conversion from RGB (inRGB) to HSV (outH, outS, outV)
// Launched with 2D gri
bool StudentWork1::isImplemented() const {
	return false;
}

__global__ void rgb2hsv(const uchar3 *const inRGB, const int width, const int height,
				float *const outH, float *const outS, float *const outV ) {
	
}

// Conversion from HSV (inH, inS, inV) to RGB (outRGB)
// Launched with 2D grid
__global__ void hsv2rgb(const float *const inH, const float *const inS, const float *const inV, 
				const int width, const int height, uchar3 *const outRGB ) {	
	
}

void StudentWork1::rgb2hsv(
	const thrust::device_vector<uchar3>&rgb,
	thrust::device_vector<float>&H,
	thrust::device_vector<float>&S,
	thrust::device_vector<float>&V
)
{

}


void StudentWork1::hsv2rgb(
	const thrust::device_vector<float>&H,
	const thrust::device_vector<float>&S,
	const thrust::device_vector<float>&V,
	thrust::device_vector<uchar3>&RGB
) 
{
}
